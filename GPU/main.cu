#include <chrono>

#include "linear.h"
#include "relu.h"
#include "train.h"
#include "../data/read_csv.h"


int main(){
    std::chrono::steady_clock::time_point begin, end;

    int tbs = 100000, n_in = 50, n_epochs = 3;
    int n_hidden = n_in/2;
    int n_out = 1;

    float *inp, *targ;  
    hipMallocManaged(&inp, tbs*n_in*sizeof(float));
    hipMallocManaged(&targ, (tbs+1)*sizeof(float));// declare 1 element more due to the fact that it is used to store loss computation result

    begin = std::chrono::steady_clock::now();
    read_csv(inp, "../data/x.csv");
    read_csv(targ, "../data/y.csv");
    end = std::chrono::steady_clock::now();
    std::cout << "Data reading time: " << (std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count())/1000000.0f << std::endl;
    
    int bs=5;
    Linear_GPU* lin1 = new Linear_GPU(bs, n_in, n_hidden);
    ReLU_GPU* relu1 = new ReLU_GPU(bs*n_hidden);
    Linear_GPU* lin2 = new Linear_GPU(bs, n_hidden, n_out);
    std::vector<Module*> layers = {lin1, relu1, lin2};

    Sequential_GPU seq(layers);
    
    begin = std::chrono::steady_clock::now();
    train_gpu(seq,inp, targ, bs, n_in,n_out, n_epochs);
    end = std::chrono::steady_clock::now();
    std::cout << "Training time: " << (std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count())/1000000.0f << std::endl;

    return 0;
}
